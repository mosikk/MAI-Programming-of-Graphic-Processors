
#include <hip/hip_runtime.h>
#include <iostream>
 
#define CSC(call) \
do { \
	hipError_t status = call; \
	if (status != hipSuccess) { \
		fprintf(stderr, "ERROR is %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status)); \
		exit(0); \
	} \
} while(0)

__global__ void kernel(double *arr1, double *arr2, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
 
	while (idx < n) {
		arr1[idx] -= arr2[idx];
		idx += offset;
	}
}

 
int main() {
    int n;
    std::cin >> n;
 
    double *arr1 = new double[n];
    double *arr2 = new double[n];
	for(int i = 0; i < n; i++) {
        std::cin >> arr1[i];
    }
    for(int i = 0; i < n; i++) {
        std::cin >> arr2[i];
    }
    
	double *dev_arr1, *dev_arr2;
    CSC(hipMalloc(&dev_arr1, sizeof(double) * n));
	CSC(hipMemcpy(dev_arr1, arr1, sizeof(double) * n, hipMemcpyHostToDevice));
 
    CSC(hipMalloc(&dev_arr2, sizeof(double) * n));
	CSC(hipMemcpy(dev_arr2, arr2, sizeof(double) * n, hipMemcpyHostToDevice));

	kernel<<<32, 32>>>(dev_arr1, dev_arr2, n);

    CSC(hipDeviceSynchronize());
	CSC(hipGetLastError());
 
    CSC(hipMemcpy(arr1, dev_arr1, sizeof(double) * n, hipMemcpyDeviceToHost));
    
	for(int i = 0; i < n; ++i) {
		std::cout << arr1[i] << " ";
    }
	std::cout << "\n";
 
    CSC(hipFree(dev_arr1));
    CSC(hipFree(dev_arr2));
 
    delete[] arr1;
    delete[] arr2;
 
	return 0;
}
